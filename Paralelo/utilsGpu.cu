#include "utilsGpu.cuh"

void *GPU_allocateMemory(unsigned long long int nbOfBytes)
{
    //std::cout << "----> " << nbOfBytes << " ------ " << checkGpuMemFree() << std::endl;
    void *ptr;
    handleError(hipMalloc((void **)&ptr, nbOfBytes));
    return ptr;
}

void GPU_setMemory(void *hostPointer, void *devicePointer, unsigned long long int nbOfBytes)
{
    handleError(hipMemcpy(devicePointer, hostPointer, nbOfBytes, hipMemcpyHostToDevice));
}

void GPU_getMemory(void *hostPointer, void *devicePointer, unsigned long long int nbOfBytes)
{
    handleError(hipMemcpy(hostPointer, devicePointer, nbOfBytes, hipMemcpyDeviceToHost));
}

// reset = memset
void GPU_resetMemory(void *devicePointer, unsigned long long int nbOfBytes)
{
    handleError(hipMemset(devicePointer, 0, nbOfBytes));
}

void GPU_freeMemory(void *memoryPointer)
{
    if (memoryPointer != NULL)
        handleError(hipFree(memoryPointer));
}