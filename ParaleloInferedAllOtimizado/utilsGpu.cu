#include "utilsGpu.cuh"

void *GPU_allocateMemory(unsigned long long int nbOfBytes)
{
    // std::cout << "----> " << nbOfBytes << " ------ " << checkGpuMemFree() << std::endl;
    void *ptr;
    handleError(hipMalloc((void **)&ptr, nbOfBytes));
    return ptr;
}

void GPU_setMemory(void *hostPointer, void *devicePointer, unsigned long long int nbOfBytes)
{
    handleError(hipMemcpy(devicePointer, hostPointer, nbOfBytes, hipMemcpyHostToDevice));
}

void GPU_getMemory(void *hostPointer, void *devicePointer, unsigned long long int nbOfBytes)
{
    handleError(hipMemcpy(hostPointer, devicePointer, nbOfBytes, hipMemcpyDeviceToHost));
}

void GPU_resetMemory(void *devicePointer, unsigned long long int nbOfBytes)
{
    handleError(hipMemset(devicePointer, 0, nbOfBytes));
}

void GPU_freeMemory(void *memoryPointer)
{
    if (memoryPointer != NULL)
        handleError(hipFree(memoryPointer));
}

uint checkGpuMemFree() // in MB
{
    float free_m;
    size_t free_t, total_t;
    hipMemGetInfo(&free_t, &total_t);
    // std::cout << "( " << free_t/1048576.0 << "  ---  " << total_t/1048576.0 << " )" << std::endl;
    // std::cout << "( " << free_t << "  ---  " << total_t << " )" << std::endl;
    free_m = free_t / 1048576.0;
    return (uint)free_m;
}