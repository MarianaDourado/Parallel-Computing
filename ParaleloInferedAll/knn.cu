#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "utilsGpu.cuh"
#include "utilsHost.hpp"

using namespace std;

#define TRAIN_FILE "../train.txt"
#define TEST_FILE "../validation.txt"
#define INF 1e9

void readTrainFile(string path, int& nbPoints, int& sizePoint, double*& points, int*& labels, map<string, int>& labelToIdx, map<int, string>& idxToLabel){
    ifstream file(path);
    if (!file.is_open()) {
        cerr << "Erro ao abrir o arquivo!" << std::endl;
        return;
    }

    double val;
    string label;

    file >> nbPoints >> sizePoint;

    points = (double*) allocHostMemory(nbPoints * sizePoint * sizeof(double));
    labels = (int*) allocHostMemory(nbPoints * sizeof(int));

    for (int i = 0; i < nbPoints; i++){
        for (int j = 0; j < sizePoint; j++){
            file >> val;
            points[i * sizePoint + j] = val;
        }
        file >> label;
        if (labelToIdx.count(label) == 0){
            idxToLabel[labelToIdx.size()] = label;
            labelToIdx[label] = labelToIdx.size();
        }
        labels[i] = labelToIdx[label];
    }

    file.close();
}

pair<double*, int*> setPointsGpu(int nbPoints, int sizePoint, double* points, int* labels){
    double* device_points = (double*) GPU_allocateMemory(nbPoints * sizePoint * sizeof(double));
    int* device_labels = (int*) GPU_allocateMemory(nbPoints * sizeof(int));

    GPU_setMemory(points, device_points, nbPoints * sizePoint * sizeof(double));
    GPU_setMemory(labels, device_labels, nbPoints * sizeof(int));

    return {device_points, device_labels};
}

__device__ double calcDist(double* pt1, double* pt2, int size){
    double dist = 0;
    for (int i = 0; i < size; i++){
        dist += (pt1[i] - pt2[i]) * (pt1[i] - pt2[i]);
    }
    return dist;
}

__global__ void KERNEL_compute_distances(double *pt, int K, int nbPoints, int sizePoint, double* kDists, int* kLabels, double* points, int* labels){
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId >= nbPoints)
        return;

    int loopStride = gridDim.x * blockDim.x;
    int start = threadId * K;

    for (int i = 0; i < K; i++){
        kDists[start + i] = INF;
        kLabels[start + i] = -1;
    }

    double dist;
    int pos;
    for (int i = threadId; i < nbPoints; i += loopStride){
        dist = calcDist(pt, &points[i * sizePoint], sizePoint);
        pos = -1;
        for (int j = K-1; j >= 0; j--){
            if (dist < kDists[start + j]){
                pos = j;
                if (j > 0){
                    kDists[start + j] = kDists[start + j - 1];
                    kLabels[start + j] = kLabels[start + j - 1];
                }
            }
            else{
                break;
            }
        }
        if (pos != -1){
            kDists[start + pos] = dist;
            kLabels[start + pos] = labels[i];
        }
        //for (int j = 0; j < K; j++){
        //    printf("%d %d -> %lf : %d\n", threadId, j, kDists[start + j], kLabels[start + j]);
        //}
    }
}

int applyKnn(double* pt, int K, int nbPoints, int sizePoint, double* device_points, int* device_labels, int nbThreadsPerBlock, int nbBlocks){
    int classe = -1;
    int nbThreads = nbThreadsPerBlock * nbBlocks;

    double* device_pt = (double*) GPU_allocateMemory(sizePoint * sizeof(double));
    GPU_setMemory(pt, device_pt, sizePoint * sizeof(double));

    double* device_kDists = (double*) GPU_allocateMemory(nbThreads * K * sizeof(double));
    int* device_kLabels = (int*) GPU_allocateMemory(nbThreads * K * sizeof(int));

    KERNEL_compute_distances<<<nbBlocks, nbThreadsPerBlock>>>(device_pt, K, nbPoints, sizePoint, device_kDists, device_kLabels, device_points, device_labels);
    handleError(hipStreamSynchronize(0));

    double* kDists = (double*) allocHostMemory(nbThreads * K * sizeof(double));
    GPU_getMemory(kDists, device_kDists, nbThreads * K * sizeof(double));

    int* kLabels = (int*) allocHostMemory(nbThreads * K * sizeof(int));
    GPU_getMemory(kLabels, device_kLabels, nbThreads * K * sizeof(int));

    vector<pair<double, int>> distsAndLabels;
    unordered_map<int, int> nbOfLabels;
    for (int i = 0; i < nbThreads * K; i++){
        distsAndLabels.push_back({kDists[i], kLabels[i]});
    }
    sort(distsAndLabels.begin(), distsAndLabels.end());
    for (int i = 0; i < K; i++){
        if (distsAndLabels[i].second != -1){
            nbOfLabels[distsAndLabels[i].second]++;
        }
    }

    for (auto [label, nb] : nbOfLabels){
        if (nbOfLabels[classe] < nb){
            classe = label;
        }
    }

    freeHostMemory(kDists);
    freeHostMemory(kLabels);    
    GPU_freeMemory(device_pt);
    GPU_freeMemory(device_kDists);
    GPU_freeMemory(device_kLabels);

    return classe;
}


int main(int argc, char *argv[])
{
    /* --------------- Obtendo o valor de K -------------- */
    if (argc != 4) {
        cout << "The number of parameters must be 3.";
        return 0;
    }
    int K = std::atoi(argv[1]);
    int nbThreadsPerBlock = atoi(argv[2]);
    int nbBlocks = atoi(argv[3]);

    /* --------------- Definindo algumas variáveis-------------- */
    clock_t start, end, startMemo, endMemo;
    map<string, int> labelToIdx = {{"-1", -1}};
    map<int, string> idxToLabel = {{-1, "-1"}};

    /* --------------- Lendo dados de treino -------------- */
    int nbPoints, sizePoint;
    double* points;
    int* labels;

    readTrainFile(TRAIN_FILE, nbPoints, sizePoint, points, labels, labelToIdx, idxToLabel);


    /* --------------- Lendo dados de teste -------------- */
    int nbPointsTest, sizePointTest;
    double *pointsTest;
    int *labelsTest;

    readTrainFile(TEST_FILE, nbPointsTest, sizePointTest, pointsTest, labelsTest, labelToIdx, idxToLabel);

    if (sizePoint != sizePointTest){
        cout << "Train and test points does not have the same dimensions.\n";
        return 0;
    }

    /* --------------- Definindo os pontos na GPU -------------- */
    startMemo = clock();
    auto [device_points, device_labels] = setPointsGpu(nbPoints, sizePoint, points, labels);
    endMemo = clock();

    /* --------------- Testando -------------- */
    double totalTime = 0, currTime;
    for (int i = 0; i < nbPointsTest; i++){
        //cout << "Test " << i << "\n";

        /* --------------- Escolhendo um ponto para testar -------------- */
        double *pt = &pointsTest[i * sizePointTest];
        int label = labelsTest[i];

        ///* --------------- Fazendo a inferência do ponto -------------- */
        start = clock();
        int inferedLabel = applyKnn(pt, K, nbPoints, sizePoint, device_points, device_labels, nbThreadsPerBlock, nbBlocks);
        end = clock();
        cout << "INFERED LABEL: " << idxToLabel[inferedLabel] << "\n";
        cout << "CORRECT CLASS: " << idxToLabel[label] << "\n";
        currTime = time_in_ms(start, end);
        cout << "TIME: " << currTime << "\n";
        totalTime += currTime;
    }

    cout <<"TOTAL TIME: " << totalTime << "\n";
    cout << "TIME FOR MEMORY: " << time_in_ms(startMemo, endMemo) << "\n";

    freeHostMemory(points);
    freeHostMemory(labels);
    GPU_freeMemory(device_points);
    GPU_freeMemory(device_labels);
    return 0;
}